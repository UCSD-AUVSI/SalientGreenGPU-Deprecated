#include "hip/hip_runtime.h"

#include <SalientGreenGPU/Cuda/dcAdjust.H>
  
__global__ void minMaxHelper( cv::gpu::DevMem2D_<float> const mat, float thresh, float * sum, int * count )
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;

	if( x > 0 && x < mat.cols - 1 && y > 0 && y < mat.rows - 1 )
	{
		const float value = mat.ptr( y )[x];
		const float neighborLeft = mat.ptr( y )[x - 1];
		const float neighborUp = mat.ptr( y - 1 )[x];
		const float neighborDown = mat.ptr( y + 1 )[x];
		const float neighborRight = mat.ptr( y )[x + 1];

		if( value >= thresh &&
				value > neighborLeft &&
				value > neighborUp &&
				value > neighborDown &&
				value >= neighborRight )
		{
			*sum += value;
			*count = *count + 1;
		}
	}
}
__global__ void absFC2( cv::gpu::DevMem2D_<myfloat2> mat )
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;

	if( x < mat.cols && y < mat.rows )
	{
		mat.ptr( y )[x].x = fabsf( mat.ptr( y )[x].x );
		mat.ptr( y )[x].y = fabsf( mat.ptr( y )[x].y );
	}
}

__global__ void maxFC2( cv::gpu::DevMem2D_<myfloat2> mat, float value )
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;

	if( x < mat.cols && y < mat.rows )
	{
		mat.ptr( y )[x].x = fmaxf( mat.ptr( y )[x].x, value );
		mat.ptr( y )[x].y = fmaxf( mat.ptr( y )[x].y, value );
	}
}

//! Adds to the DC offset for a matrix
/*! @param[in] mat The CUDA device memory for the matrix
	  @param[in] value The value to add to the DC component */
__global__ void dcAdjust( cv::gpu::DevMem2D_<myfloat2> mat, float value )
{
	mat.ptr( 0 )[0].x += value;
}

//! computes c = a + b
__global__ void addFC2( cv::gpu::DevMem2D_<myfloat2> const a, cv::gpu::DevMem2D_<myfloat2> const b,
		cv::gpu::DevMem2D_<myfloat2> c )
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;

	if( x < a.cols && y < a.rows )
	{
		c.ptr( y )[x].x = a.ptr( y )[x].x + b.ptr( y )[x].x;
		c.ptr( y )[x].y = a.ptr( y )[x].y + b.ptr( y )[x].y;
	}
}

__global__ void addAndZeroFC2( cv::gpu::DevMem2D_<myfloat2> mat, float value )
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;

	if( x < mat.cols && y < mat.rows )
	{
		mat.ptr( y )[x].x += value; fmaxf( mat.ptr( y )[x].x, value );
		mat.ptr( y )[x].y = 0.0f;
	}
}

//! computes c = a - b
__global__ void subFC2( cv::gpu::DevMem2D_<myfloat2> const a, cv::gpu::DevMem2D_<myfloat2> const b,
		cv::gpu::DevMem2D_<myfloat2> c )
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;

	if( x < a.cols && y < a.rows )
	{
		c.ptr( y )[x].x = a.ptr( y )[x].x - b.ptr( y )[x].x;
		c.ptr( y )[x].y = a.ptr( y )[x].y - b.ptr( y )[x].y;
	}

}
//! computes c = a * b
__global__ void mulFC2( cv::gpu::DevMem2D_<myfloat2> const a, cv::gpu::DevMem2D_<myfloat2> const b,
		cv::gpu::DevMem2D_<myfloat2> c )
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;

	if( x < a.cols && y < a.rows )
	{
		c.ptr( y )[x].x = a.ptr( y )[x].x * b.ptr( y )[x].x;
		c.ptr( y )[x].y = a.ptr( y )[x].y * b.ptr( y )[x].y;
	}

}

//! computes c = a * b
__global__ void mulValueFC2( cv::gpu::DevMem2D_<myfloat2> const a, float const b,
		cv::gpu::DevMem2D_<myfloat2> c )
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;

	if( x < a.cols && y < a.rows )
	{
		c.ptr( y )[x].x = a.ptr( y )[x].x * b;
		c.ptr( y )[x].y = a.ptr( y )[x].y * b;
	}

}

static inline int divUp( int total, int grain )
{
	return ( total + grain - 1 ) / grain;
}

void addFC2Wrapper( cv::gpu::DevMem2D_<myfloat2> const a, cv::gpu::DevMem2D_<myfloat2> const b,
		cv::gpu::DevMem2D_<myfloat2> c )
{
	dim3 threads( 16, 16, 1 );
	dim3 grids( 1, 1, 1 );

	grids.x = divUp( a.cols, threads.x );
	grids.y = divUp( a.rows, threads.y );

	addFC2<<< grids, threads >>>( a, b, c );
}

void addAndZeroFC2Wrapper( cv::gpu::DevMem2D_<myfloat2> mat, float value )
{
	dim3 threads( 16, 16, 1 );
	dim3 grids( 1, 1, 1 );

	grids.x = divUp( mat.cols, threads.x );
	grids.y = divUp( mat.rows, threads.y );

	addAndZeroFC2<<< grids, threads >>>( mat, value );
}
void subFC2Wrapper( cv::gpu::DevMem2D_<myfloat2> const a, cv::gpu::DevMem2D_<myfloat2> const b,
		cv::gpu::DevMem2D_<myfloat2> c )
{
	dim3 threads( 16, 16, 1 );
	dim3 grids( 1, 1, 1 );

	grids.x = divUp( a.cols, threads.x );
	grids.y = divUp( a.rows, threads.y );

	subFC2<<< grids, threads >>>( a, b, c );
}

void mulFC2Wrapper( cv::gpu::DevMem2D_<myfloat2> const a, cv::gpu::DevMem2D_<myfloat2> const b,
		cv::gpu::DevMem2D_<myfloat2> c )
{
	dim3 threads( 16, 16, 1 );
	dim3 grids( 1, 1, 1 );

	grids.x = divUp( a.cols, threads.x );
	grids.y = divUp( a.rows, threads.y );

	mulFC2<<< grids, threads >>>( a, b, c );
}

void mulValueFC2Wrapper( cv::gpu::DevMem2D_<myfloat2> const a, float const b,
		cv::gpu::DevMem2D_<myfloat2> c )
{
	dim3 threads( 16, 16, 1 );
	dim3 grids( 1, 1, 1 );

	grids.x = divUp( a.cols, threads.x );
	grids.y = divUp( a.rows, threads.y );

	mulValueFC2<<< grids, threads >>>( a, b, c );
}

void dcAdjustWrapper( cv::gpu::DevMem2D_<myfloat2> mat, float value )
{
	dim3 threads( 16, 16, 1 );
	dim3 grids( 1, 1, 1 );

	grids.x = divUp( mat.cols, threads.x );
	grids.y = divUp( mat.rows, threads.y );

	dcAdjust<<< grids, threads >>>( mat, value );
}


void absFC2Wrapper( cv::gpu::DevMem2D_<myfloat2> mat )
{
	dim3 threads( 16, 16, 1 );
	dim3 grids( 1, 1, 1 );

	grids.x = divUp( mat.cols, threads.x );
	grids.y = divUp( mat.rows, threads.y );

	absFC2<<< grids, threads >>>( mat );
}

void maxFC2Wrapper( cv::gpu::DevMem2D_<myfloat2> mat, float value )
{
	dim3 threads( 16, 16, 1 );
	dim3 grids( 1, 1, 1 );

	grids.x = divUp( mat.cols, threads.x );
	grids.y = divUp( mat.rows, threads.y );

	maxFC2<<< grids, threads >>>( mat, value );
}

void minMaxHelperWrapper( cv::gpu::DevMem2D_<float> const mat, float thresh, float & sum, int & count )
{
	dim3 threads( 16, 16, 1 );
	dim3 grids( 1, 1, 1 );

	grids.x = divUp( mat.cols, threads.x );
	grids.y = divUp( mat.rows, threads.y );

	float * dSum;
	int * dCount;
	hipMalloc(&dSum, sizeof(float));
	hipMalloc(&dCount, sizeof(int));

	minMaxHelper<<< grids, threads >>>( mat, thresh, dSum, dCount );
	hipMemcpy( &sum, dSum, sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( &count, dCount, sizeof(int), hipMemcpyDeviceToHost );
	hipFree( dCount );
	hipFree( dSum );
}
